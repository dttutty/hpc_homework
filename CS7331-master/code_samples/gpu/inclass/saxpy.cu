
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VAL_RANGE 1023

double mysecond() {
  struct timeval tp;
  struct timezone tzp;
  gettimeofday(&tp,&tzp);
  return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void saxpy_cpu(int n, float s, float* x, float* y) {
  for(int i = 0; i < n; i++) 
    y[i] = s * x[i] + y[i];
}


__global__ void  saxpy(int n, float s, float* x, float* y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = s * x[i] + y[i];
}

int main(int argc, char *argv[]) {

  float *x, *y;
  float *result;

  if (argc < 2) {
    printf("usage: \n");
    printf("       ./saxpy <size>\n");
    exit(0);
  }

  long long N = atoi(argv[1]);
  int i;

  x = (float *) malloc(sizeof(float) * N);
  y = (float *) malloc(sizeof(float) * N);

  result = (float *) malloc(sizeof(float) * N);

  for (i = 0; i < N; i++) {
    x[i] = rand() / (float) (RAND_MAX/VAL_RANGE);
    y[i] = rand() / (float) (RAND_MAX/VAL_RANGE);
  }

  float* dev_x;
  float* dev_y;
  hipMalloc(&dev_x, N * sizeof(float));
  hipMalloc(&dev_y, N * sizeof(float));

  float s = 2.3;

  hipMemcpy(dev_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  int blocksize = 1024;
  int numblocks = N/blocksize; // assume N is evenly divisible by block size
  saxpy<<<numblocks, blocksize>>>(N, s, dev_x, dev_y);

  hipMemcpy(result, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

  // verify

  saxpy_cpu(N, s, x, y);
  if (y[17] == result[17]) 
    printf("Passed %3.4f.\n", result[17]);

  hipDeviceReset();
  return 0;
}

